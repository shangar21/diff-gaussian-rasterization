#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
template <typename floatp, typename vecp3>
__device__ vecp3 computeColorFromSH(int idx, int deg, int max_coeffs, const vecp3* means, vecp3 campos, const floatp* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	vecp3 pos = means[idx];
	vecp3 dir = pos - campos;
	dir = dir / glm::length(dir);

	vecp3* sh = ((vecp3*)shs) + idx * max_coeffs;
	vecp3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		floatp x = dir.x;
		floatp y = dir.y;
		floatp z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			floatp xx = x * x, yy = y * y, zz = z * z;
			floatp xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
template <typename floatp, typename floatp3>
__device__ floatp3 computeCov2D(const floatp3& mean, floatp focal_x, floatp focal_y, floatp tan_fovx, floatp tan_fovy, const floatp* cov3D, const floatp* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	floatp3 t = transformPoint4x3(mean, viewmatrix);

	const floatp limx = 1.3f * tan_fovx;
	const floatp limy = 1.3f * tan_fovy;
	const floatp txtz = t.x / t.z;
	const floatp tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::tmat3x3<floatp> J = glm::tmat3x3<floatp>(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::tmat3x3<floatp> W = glm::tmat3x3<floatp>(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::tmat3x3<floatp> T = W * J;

	glm::tmat3x3<floatp> Vrk = glm::tmat3x3<floatp>(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::tmat3x3<floatp> cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { floatp(cov[0][0]), floatp(cov[0][1]), floatp(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
template <typename floatp, typename vec3p, typename vec4p>
__device__ void computeCov3D(const vec3p scale, floatp mod, const vec4p rot, floatp* cov3D)
{
	// Create scaling matrix
	glm::tmat3x3<floatp> S = glm::tmat3x3<floatp>(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	vec4p q = rot;// / glm::length(rot);
	floatp r = q.x;
	floatp x = q.y;
	floatp y = q.z;
	floatp z = q.w;

	// Compute rotation matrix from quaternion
	glm::tmat3x3<floatp> R = glm::tmat3x3<floatp>(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::tmat3x3<floatp> M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::tmat3x3<floatp> Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template <int C, typename floatp, typename floatp2, typename floatp3, typename floatp4, typename vec3p, typename vec4p>
__global__ void preprocessCUDA(int P, int D, int M,
	const floatp* orig_points,
	const vec3p* scales,
	const floatp scale_modifier,
	const vec4p* rotations,
	const floatp* opacities,
	const floatp* shs,
	bool* clamped,
	const floatp* cov3D_precomp,
	const floatp* colors_precomp,
	const floatp* viewmatrix,
	const floatp* projmatrix,
	const vec3p* cam_pos,
	const int W, int H,
	const floatp tan_fovx, floatp tan_fovy,
	const floatp focal_x, floatp focal_y,
	int* radii,
	floatp2* points_xy_image,
	floatp* depths,
	floatp* cov3Ds,
	floatp* rgb,
	floatp4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	floatp3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	floatp3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	floatp4 p_hom = transformPoint4x4(p_orig, projmatrix);
	floatp p_w = 1.0f / (p_hom.w + 0.0000001f);
	floatp3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const floatp* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D<floatp, vec3p, vec4p>(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	floatp3 cov = computeCov2D<floatp, floatp3>(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	floatp det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	floatp det_inv = 1.f / det;
	floatp3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	floatp mid = 0.5f * (cov.x + cov.z);
	floatp lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	floatp lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	floatp my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	floatp2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		vec3p result = computeColorFromSH<floatp, vec3p>(idx, D, M, (vec3p*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one floatp4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS, typename floatp, typename floatp2, typename floatp4>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const floatp2* __restrict__ points_xy_image,
	const floatp* __restrict__ features,
	const floatp4* __restrict__ conic_opacity,
	floatp* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const floatp* __restrict__ bg_color,
	floatp* __restrict__ out_color)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	floatp2 pixf = { (floatp)pix.x, (floatp)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ floatp2 collected_xy[BLOCK_SIZE];
	__shared__ floatp4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	floatp T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	floatp C[CHANNELS] = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			floatp2 xy = collected_xy[j];
			floatp2 d = { xy.x - pixf.x, xy.y - pixf.y };
			floatp4 con_o = collected_conic_opacity[j];
			floatp power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			floatp alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			floatp test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

template <typename floatp, typename floatp2, typename floatp4>
void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const floatp2* means2D,
	const floatp* colors,
	const floatp4* conic_opacity,
	floatp* final_T,
	uint32_t* n_contrib,
	const floatp* bg_color,
	floatp* out_color)
{
	renderCUDA<NUM_CHANNELS, floatp, floatp2, floatp4> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color);
}

template <typename floatp, typename floatp2, typename floatp3, typename floatp4, typename vec3p, typename vec4p>
void FORWARD::preprocess(int P, int D, int M,
	const floatp* means3D,
	const vec3p* scales,
	const floatp scale_modifier,
	const vec4p* rotations,
	const floatp* opacities,
	const floatp* shs,
	bool* clamped,
	const floatp* cov3D_precomp,
	const floatp* colors_precomp,
	const floatp* viewmatrix,
	const floatp* projmatrix,
	const vec3p* cam_pos,
	const int W, int H,
	const floatp focal_x, floatp focal_y,
	const floatp tan_fovx, floatp tan_fovy,
	int* radii,
	floatp2* means2D,
	floatp* depths,
	floatp* cov3Ds,
	floatp* rgb,
	floatp4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS, floatp, floatp2, floatp3, floatp4, vec3p, vec4p> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}

template void FORWARD::render<float, float2, float4>(
		const dim3 grid, dim3 block,
		const uint2* ranges,
		const uint32_t* point_list,
		int W, int H,
		const float2* points_xy_image,
		const float* features,
		const float4* conic_opacity,
		float* final_T,
		uint32_t* n_contrib,
		const float* bg_color,
		float* out_color
	);

template void FORWARD::preprocess<float, float2, float3, float4, glm::vec3, glm::vec4>(
		int P, int D, int M,
		const float* orig_points,
		const glm::vec3* scales,
		const float scale_modifier,
		const glm::vec4* rotations,
		const float* opacities,
		const float* shs,
		bool* clamped,
		const float* cov3D_precomp,
		const float* colors_precomp,
		const float* viewmatrix,
		const float* projmatrix,
		const glm::vec3* cam_pos,
		const int W, int H,
		const float focal_x, float focal_y,
		const float tan_fovx, float tan_fovy,
		int* radii,
		float2* points_xy_image,
		float* depths,
		float* cov3Ds,
		float* colors,
		float4* conic_opacity,
		const dim3 grid,
		uint32_t* tiles_touched,
		bool prefiltered
	);

